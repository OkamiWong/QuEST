#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <functional>
#include <map>
#include <vector>

#include "../../../dependencies/optimize-cuda-memory-usage-v1/public/memopt.hpp"
#include "QuEST.h"
#include "QuEST_gpu_common.h"
#include "QuEST_internal.h"
#include "QuEST_precision.h"
#include "QuEST_validation.h"
#include "mt19937ar.h"

// General utilities
template <typename T>
void __check(T result, char const* const func, const char* const file, int const line) {
  if (result) {
    fprintf(stderr, "CUDA error at %s:%d code=%d \"%s\" \n", file, line, static_cast<unsigned int>(result), func);
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(val) __check((val), #val, __FILE__, __LINE__)

typedef long long StateVecIndex_t;

// Memopt wrappers
namespace memopt_adapter {

template <typename T>
void allocateShardAndRegister(T** p, size_t s) {
  checkCudaErrors(hipMalloc(p, s));
  memopt::registerManagedMemoryAddress(*p, s);
  memopt::registerApplicationInput(*p);
  memopt::registerApplicationOutput(*p);
}

typedef std::function<void(Qureg, hipStream_t)> Task;

std::vector<Task> tasks;

void registerAndExecuteTask(
  std::vector<StateVecIndex_t> inputShardIndices,
  std::vector<StateVecIndex_t> outputShardIndices,
  Task task,
  Qureg qureg,
  hipStream_t stream
) {
  auto taskId = tasks.size();
  tasks.push_back(task);

  std::vector<void*> inputs, outputs;
  for (auto i : inputShardIndices) {
    inputs.push_back(qureg.deviceStateVecShards[i].real);
    inputs.push_back(qureg.deviceStateVecShards[i].imag);
  }
  for (auto i : outputShardIndices) {
    outputs.push_back(qureg.deviceStateVecShards[i].real);
    outputs.push_back(qureg.deviceStateVecShards[i].imag);
  }

  memopt::annotateNextTask(taskId, inputs, outputs, stream);
  task(qureg, stream);
}

template <typename T>
void tryUpdatingAddress(T*& oldAddress, const std::map<void*, void*>& addressUpdateMap) {
  if (addressUpdateMap.count(oldAddress) > 0) {
    oldAddress = (T*)addressUpdateMap.at(oldAddress);
  }
}

void executeRandomTask(Qureg qureg, int taskId, std::map<void*, void*> addressUpdateMap, hipStream_t stream) {
  for (StateVecIndex_t i = 0; i < qureg.numShards; i++) {
    tryUpdatingAddress(qureg.deviceStateVecShards[i].real, addressUpdateMap);
    tryUpdatingAddress(qureg.deviceStateVecShards[i].imag, addressUpdateMap);
  }
  tasks[taskId](qureg, stream);
}

}  // namespace memopt_adapter

// QuEST speicifc definitions
constexpr int MAX_NUM_QUBITS = 64;
constexpr int MAX_NUM_PHASE_FUNC_OVERRIDES = 8;

__forceinline__ __device__ int getBit(StateVecIndex_t num, int index) {
  return (num >> index) & 1;
}

__forceinline__ __host__ __device__ int extractBit(const int locationOfBitFromRight, const StateVecIndex_t theEncodedNumber) {
  return (theEncodedNumber & (1LL << locationOfBitFromRight)) >> locationOfBitFromRight;
}

__forceinline__ __host__ __device__ StateVecIndex_t flipBit(const StateVecIndex_t number, const int index) {
  return (number ^ (1LL << index));
}

__forceinline__ __host__ __device__ StateVecIndex_t insertZeroBit(const StateVecIndex_t number, const int index) {
  StateVecIndex_t left, right;
  left = (number >> index) << index;
  right = number - left;
  return (left << 1) ^ right;
}

__forceinline__ __host__ __device__ StateVecIndex_t insertTwoZeroBits(const StateVecIndex_t number, const int bit1, const int bit2) {
  int small = (bit1 < bit2) ? bit1 : bit2;
  int big = (bit1 < bit2) ? bit2 : bit1;
  return insertZeroBit(insertZeroBit(number, small), big);
}

__forceinline__ __host__ __device__ StateVecIndex_t getGlobalIndex(Qureg* qureg, StateVecIndex_t index) {
  return index >> qureg->numLocalBits;
}

__forceinline__ __host__ __device__ StateVecIndex_t getLocalIndex(Qureg* qureg, StateVecIndex_t index) {
  return index & ((1 << qureg->numLocalBits) - 1);
}

__forceinline__ __host__ __device__ void splitIndex(Qureg* qureg, StateVecIndex_t index, StateVecIndex_t* globalIndex, StateVecIndex_t* localIndex) {
  *globalIndex = getGlobalIndex(qureg, index);
  *localIndex = getLocalIndex(qureg, index);
}

qreal statevec_getRealAmp(Qureg qureg, StateVecIndex_t index) {
  StateVecIndex_t globalIndex, localIndex;
  splitIndex(&qureg, index, &globalIndex, &localIndex);

  qreal el = 0;
  hipMemcpy(
    &el,
    &(qureg.deviceStateVecShards[globalIndex].real[localIndex]),
    sizeof(qreal),
    hipMemcpyDefault
  );
  return el;
}

qreal statevec_getImagAmp(Qureg qureg, StateVecIndex_t index) {
  StateVecIndex_t globalIndex, localIndex;
  splitIndex(&qureg, index, &globalIndex, &localIndex);

  qreal el = 0;
  hipMemcpy(
    &el,
    &(qureg.deviceStateVecShards[globalIndex].imag[localIndex]),
    sizeof(qreal),
    hipMemcpyDefault
  );
  return el;
}

void statevec_initZeroState(Qureg qureg) {
  // Set all amps to zero
  for (StateVecIndex_t i = 0; i < qureg.numShards; i++) {
    checkCudaErrors(hipMemset(qureg.deviceStateVecShards[i].real, 0, qureg.numAmpsPerShard * sizeof(qreal)));
    checkCudaErrors(hipMemset(qureg.deviceStateVecShards[i].imag, 0, qureg.numAmpsPerShard * sizeof(qreal)));
  }

  // Set |000...000>'s amp to 1
  qreal one = 1, zero = 0;
  checkCudaErrors(hipMemcpy(&qureg.deviceStateVecShards[0].real[0], &one, sizeof(qreal), hipMemcpyDefault));
  checkCudaErrors(hipMemcpy(&qureg.deviceStateVecShards[0].imag[0], &zero, sizeof(qreal), hipMemcpyDefault));
}

__global__ void statevec_hadamardLocalBitKernel(Qureg qureg, StateVecIndex_t globalIndex, int targetQubit) {
  const StateVecIndex_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const StateVecIndex_t numTasks = qureg.numAmpsPerShard >> 1;
  if (idx >= numTasks) return;

  StateVecIndex_t indexUp = insertZeroBit(idx, targetQubit);
  StateVecIndex_t indexLo = flipBit(indexUp, targetQubit);

  qreal* stateVecReal = qureg.deviceStateVecShards[globalIndex].real;
  qreal* stateVecImag = qureg.deviceStateVecShards[globalIndex].imag;

  qreal *stateRealUp, *stateRealLo, *stateImagUp, *stateImagLo;
  stateRealUp = &stateVecReal[indexUp];
  stateImagUp = &stateVecImag[indexUp];
  stateRealLo = &stateVecReal[indexLo];
  stateImagLo = &stateVecImag[indexLo];

  qreal stateRealUpValue, stateRealLoValue, stateImagUpValue, stateImagLoValue;
  stateRealUpValue = *stateRealUp;
  stateImagUpValue = *stateImagUp;
  stateRealLoValue = *stateRealLo;
  stateImagLoValue = *stateImagLo;

  qreal factor = 1.0 / sqrt(2.0);

  *stateRealUp = factor * (stateRealUpValue + stateRealLoValue);
  *stateImagUp = factor * (stateImagUpValue + stateImagLoValue);
  *stateRealLo = factor * (stateRealUpValue - stateRealLoValue);
  *stateImagLo = factor * (stateImagUpValue - stateImagLoValue);
}

__global__ void statevec_hadamardGlobalBitKernel(Qureg qureg, StateVecIndex_t globalIndex, int targetQubit) {
  const StateVecIndex_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= qureg.numAmpsPerShard) return;

  StateVecIndex_t globalIndexUp = globalIndex;
  StateVecIndex_t globalIndexLo = globalIndex | (1 << (targetQubit - qureg.numLocalBits));

  ComplexArray* stateVecShardUp = &qureg.deviceStateVecShards[globalIndexUp];
  ComplexArray* stateVecShardLo = &qureg.deviceStateVecShards[globalIndexLo];

  qreal *stateRealUp, *stateRealLo, *stateImagUp, *stateImagLo;
  stateRealUp = &stateVecShardUp->real[idx];
  stateImagUp = &stateVecShardUp->imag[idx];
  stateRealLo = &stateVecShardLo->real[idx];
  stateImagLo = &stateVecShardLo->imag[idx];

  qreal stateRealUpValue, stateRealLoValue, stateImagUpValue, stateImagLoValue;
  stateRealUpValue = *stateRealUp;
  stateImagUpValue = *stateImagUp;
  stateRealLoValue = *stateRealLo;
  stateImagLoValue = *stateImagLo;

  qreal factor = 1.0 / sqrt(2.0);

  *stateRealUp = factor * (stateRealUpValue + stateRealLoValue);
  *stateImagUp = factor * (stateImagUpValue + stateImagLoValue);
  *stateRealLo = factor * (stateRealUpValue - stateRealLoValue);
  *stateImagLo = factor * (stateImagUpValue - stateImagLoValue);
}

void memopt_statevec_hadamard(hipStream_t stream, Qureg qureg, int targetQubit) {
  if (targetQubit < qureg.numLocalBits) {
    StateVecIndex_t numThreadsPerBlock, numBlocks;
    numThreadsPerBlock = 128;
    numBlocks = ((qureg.numAmpsPerShard >> 1) + numThreadsPerBlock - 1) / numThreadsPerBlock;

    for (StateVecIndex_t i = 0; i < qureg.numShards; i++) {
      memopt_adapter::Task task = [=](Qureg q, hipStream_t s) {
        statevec_hadamardLocalBitKernel<<<numBlocks, numThreadsPerBlock, 0, s>>>(
          q,
          i,
          targetQubit
        );
      };
      memopt_adapter::registerAndExecuteTask(
        {i},
        {i},
        task,
        qureg,
        stream
      );
    }
  } else {
    StateVecIndex_t numThreadsPerBlock, numBlocks;
    numThreadsPerBlock = 128;
    numBlocks = (qureg.numAmpsPerShard + numThreadsPerBlock - 1) / numThreadsPerBlock;

    for (StateVecIndex_t i = 0; i < (qureg.numShards >> 1); i++) {
      StateVecIndex_t globalIndexUp = insertZeroBit(i, targetQubit - qureg.numLocalBits);
      StateVecIndex_t globalIndexLo = flipBit(globalIndexUp, targetQubit - qureg.numLocalBits);
      memopt_adapter::Task task = [=](Qureg q, hipStream_t s) {
        statevec_hadamardGlobalBitKernel<<<numBlocks, numThreadsPerBlock, 0, s>>>(
          q,
          globalIndexUp,
          targetQubit
        );
      };
      memopt_adapter::registerAndExecuteTask(
        {globalIndexUp, globalIndexLo},
        {globalIndexUp, globalIndexLo},
        task,
        qureg,
        stream
      );
    }
  }
}

__forceinline__ __device__ void setMultiRegPhaseInds(
  Qureg* qureg,
  StateVecIndex_t* phaseInds, StateVecIndex_t fullIndex,
  int* qubits, int* numQubitsPerReg, int numRegs, enum bitEncoding encoding
) {
  size_t stride = qureg->numAmpsPerShard;
  size_t offset = fullIndex & ((1 << qureg->numLocalBits) - 1);

  if (encoding == UNSIGNED) {
    int flatIndex = 0;
    for (int r = 0; r < numRegs; r++) {
      phaseInds[r * stride + offset] = 0LL;
      for (int q = 0; q < numQubitsPerReg[r]; q++)
        phaseInds[r * stride + offset] += (1LL << q) * getBit(fullIndex, qubits[flatIndex++]);
    }
  } else if (encoding == TWOS_COMPLEMENT) {
    int flatIndex = 0;
    for (int r = 0; r < numRegs; r++) {
      phaseInds[r * stride + offset] = 0LL;
      for (int q = 0; q < numQubitsPerReg[r] - 1; q++)
        phaseInds[r * stride + offset] += (1LL << q) * getBit(fullIndex, qubits[flatIndex++]);

      // Use final qubit to indicate sign
      if (getBit(fullIndex, qubits[flatIndex++]) == 1)
        phaseInds[r * stride + offset] -= (1LL << (numQubitsPerReg[r] - 1));
    }
  }
}

__forceinline__ __device__ StateVecIndex_t getIndOfMultiRegPhaseOverride(
  Qureg* qureg,
  StateVecIndex_t fullIndex,
  StateVecIndex_t* phaseInds, int numRegs,
  StateVecIndex_t* overrideInds, int numOverrides
) {
  size_t stride = qureg->numAmpsPerShard;
  size_t offset = fullIndex & ((1 << qureg->numLocalBits) - 1);

  int i;
  for (i = 0; i < numOverrides; i++) {
    int found = 1;
    for (int r = 0; r < numRegs; r++) {
      if (phaseInds[r * stride + offset] != overrideInds[i * numRegs + r]) {
        found = 0;
        break;
      }
    }

    if (found)
      break;
  }

  return i;
}

__forceinline__ __device__ qreal evalNormPhaseFunc(
  StateVecIndex_t* phaseInds, size_t stride, size_t offset,
  int numRegs, enum phaseFunc phaseFuncName, qreal* params, int numParams
) {
  // determine norm
  qreal norm = 0;
  if (phaseFuncName == SCALED_INVERSE_SHIFTED_NORM) {
    for (int r = 0; r < numRegs; r++) {
      qreal dif = phaseInds[r * stride + offset] - params[2 + r];
      norm += dif * dif;
    }
  } else
    for (int r = 0; r < numRegs; r++)
      norm += phaseInds[r * stride + offset] * phaseInds[r * stride + offset];
  norm = sqrt(norm);

  // determine phase via phase function

  if (phaseFuncName == NORM)
    return norm;

  if (phaseFuncName == INVERSE_NORM)
    return (norm == 0.) ? params[0] : 1 / norm;  // smallest non-zero norm is 1

  if (phaseFuncName == SCALED_NORM)
    return params[0] * norm;

  if (
    phaseFuncName == SCALED_INVERSE_NORM || phaseFuncName == SCALED_INVERSE_SHIFTED_NORM
  )
    return (norm <= REAL_EPS) ? params[1] : params[0] / norm;  // unless shifted closer to zero
}

__forceinline__ __device__ qreal evalProductPhaseFunc(
  StateVecIndex_t* phaseInds, size_t stride, size_t offset,
  int numRegs, enum phaseFunc phaseFuncName, qreal* params, int numParams
) {
  // determine product of phase indices
  qreal prod = 1;
  for (int r = 0; r < numRegs; r++)
    prod *= phaseInds[r * stride + offset];

  // determine phase via phase function
  if (phaseFuncName == PRODUCT)
    return prod;

  if (phaseFuncName == INVERSE_PRODUCT)
    return (prod == 0.) ? params[0] : 1 / prod;  // smallest non-zero prod is +- 1

  if (phaseFuncName == SCALED_PRODUCT)
    return params[0] * prod;

  if (phaseFuncName == SCALED_INVERSE_PRODUCT)
    return (prod == 0.) ? params[1] : params[0] / prod;
}

__forceinline__ __device__ qreal evalDistancePhaseFunc(
  StateVecIndex_t* phaseInds, size_t stride, size_t offset,
  int numRegs, enum phaseFunc phaseFuncName, qreal* params, int numParams
) {
  // evaluate distance (depends on phase function)
  qreal dist = 0;
  if (phaseFuncName == SCALED_INVERSE_SHIFTED_DISTANCE) {
    for (int r = 0; r < numRegs; r += 2) {
      qreal dif = (phaseInds[r * stride + offset] - phaseInds[(r + 1) * stride + offset] - params[2 + r / 2]);
      dist += dif * dif;
    }
  } else if (phaseFuncName == SCALED_INVERSE_SHIFTED_WEIGHTED_DISTANCE) {
    for (int r = 0; r < numRegs; r += 2) {
      qreal dif = (phaseInds[r * stride + offset] - phaseInds[(r + 1) * stride + offset] - params[2 + r + 1]);
      dist += params[2 + r] * dif * dif;
    }
  } else
    for (int r = 0; r < numRegs; r += 2) {
      qreal dif = (phaseInds[(r + 1) * stride + offset] - phaseInds[r * stride + offset]);
      dist += dif * dif;
    }

  // if sqrt() arg of distance would be negative, set it to zero, to subsequently be set to the divergence param
  if (dist < 0)
    dist = 0;

  dist = sqrt(dist);

  if (phaseFuncName == DISTANCE)
    return dist;

  if (phaseFuncName == INVERSE_DISTANCE)
    return (dist == 0.) ? params[0] : 1 / dist;  // smallest non-zero dist is 1

  if (phaseFuncName == SCALED_DISTANCE)
    return params[0] * dist;

  if (
    phaseFuncName == SCALED_INVERSE_DISTANCE || phaseFuncName == SCALED_INVERSE_SHIFTED_DISTANCE || phaseFuncName == SCALED_INVERSE_SHIFTED_WEIGHTED_DISTANCE
  )
    return (dist <= REAL_EPS) ? params[1] : params[0] / dist;  // unless shifted closer
}

__forceinline__ __device__ qreal getPhaseFromParamNamedFunc(
  Qureg* qureg,
  StateVecIndex_t fullIndex,
  StateVecIndex_t* phaseInds, int numRegs,
  enum phaseFunc phaseFuncName, qreal* params, int numParams
) {
  size_t stride = qureg->numAmpsPerShard;
  size_t offset = fullIndex & ((1 << qureg->numLocalBits) - 1);

  if (
    phaseFuncName == NORM
    || phaseFuncName == INVERSE_NORM
    || phaseFuncName == SCALED_NORM
    || phaseFuncName == SCALED_INVERSE_NORM
    || phaseFuncName == SCALED_INVERSE_SHIFTED_NORM
  )
    return evalNormPhaseFunc(phaseInds, stride, offset, numRegs, phaseFuncName, params, numParams);

  if (
    phaseFuncName == PRODUCT
    || phaseFuncName == INVERSE_PRODUCT
    || phaseFuncName == SCALED_PRODUCT
    || phaseFuncName == SCALED_INVERSE_PRODUCT
  )
    return evalProductPhaseFunc(phaseInds, stride, offset, numRegs, phaseFuncName, params, numParams);

  if (
    phaseFuncName == DISTANCE
    || phaseFuncName == INVERSE_DISTANCE
    || phaseFuncName == SCALED_DISTANCE
    || phaseFuncName == SCALED_INVERSE_DISTANCE
    || phaseFuncName == SCALED_INVERSE_SHIFTED_DISTANCE
    || phaseFuncName == SCALED_INVERSE_SHIFTED_WEIGHTED_DISTANCE
  )
    return evalDistancePhaseFunc(phaseInds, stride, offset, numRegs, phaseFuncName, params, numParams);
}

__forceinline__ __device__ void applyPhaseToAmp(
  Qureg* qureg,
  StateVecIndex_t globalIndex, StateVecIndex_t localIndex,
  qreal phase, int conj
) {
  phase *= (1 - 2 * conj);
  qreal c = cos(phase);
  qreal s = sin(phase);

  qreal re, im;
  re = qureg->deviceStateVecShards[globalIndex].real[localIndex];
  im = qureg->deviceStateVecShards[globalIndex].imag[localIndex];
  qureg->deviceStateVecShards[globalIndex].real[localIndex] = re * c - im * s;
  qureg->deviceStateVecShards[globalIndex].imag[localIndex] = re * s + im * c;
}

struct ApplyParamNamedPhaseFuncOverridesParams {
  int qubits[MAX_NUM_QUBITS];
  int numQubitsPerReg[MAX_NUM_QUBITS];
  int numRegs;
  enum bitEncoding encoding;
  enum phaseFunc phaseFuncName;
  qreal params[MAX_NUM_QUBITS + 2];
  int numParams;
  StateVecIndex_t overrideInds[MAX_NUM_PHASE_FUNC_OVERRIDES];
  qreal overridePhases[MAX_NUM_PHASE_FUNC_OVERRIDES];
  int numOverrides;
  int conj;
};

__global__ void statevec_applyParamNamedPhaseFuncOverridesKernel(
  Qureg qureg,
  ApplyParamNamedPhaseFuncOverridesParams params,
  StateVecIndex_t* phaseInds,
  StateVecIndex_t globalIndex
) {
  const StateVecIndex_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= qureg.numAmpsPerShard) return;

  StateVecIndex_t fullIndex = idx + (globalIndex << qureg.numLocalBits);

  // Determine phase indices (each thread has phaseInds[numRegs] sub-array)
  setMultiRegPhaseInds(&qureg, phaseInds, fullIndex, params.qubits, params.numQubitsPerReg, params.numRegs, params.encoding);

  // Determine if this phase index has an overriden value
  StateVecIndex_t overrideCaseIndex = getIndOfMultiRegPhaseOverride(&qureg, fullIndex, phaseInds, params.numRegs, params.overrideInds, params.numOverrides);

  // Determine the phase, or the overriden one
  qreal phase = 0;
  if (overrideCaseIndex < params.numOverrides)
    phase = params.overridePhases[overrideCaseIndex];
  else
    phase = getPhaseFromParamNamedFunc(&qureg, fullIndex, phaseInds, params.numRegs, params.phaseFuncName, params.params, params.numParams);

  // Modify amp to amp * exp(i phase)
  applyPhaseToAmp(&qureg, globalIndex, idx, phase, params.conj);
}

void memopt_statevec_applyParamNamedPhaseFuncOverrides(
  hipStream_t stream,
  Qureg qureg,
  ApplyParamNamedPhaseFuncOverridesParams params
) {
  StateVecIndex_t numThreadsPerBlock, numBlocks;
  numThreadsPerBlock = 128;
  numBlocks = (qureg.numAmpsPerShard + numThreadsPerBlock - 1) / numThreadsPerBlock;

  for (StateVecIndex_t i = 0; i < qureg.numShards; i++) {
    memopt_adapter::Task task = [=](Qureg q, hipStream_t s) {
      StateVecIndex_t* d_phaseInds;
      checkCudaErrors(hipMallocAsync(&d_phaseInds, params.numRegs * q.numAmpsPerShard * sizeof *d_phaseInds, s));
      statevec_applyParamNamedPhaseFuncOverridesKernel<<<numBlocks, numThreadsPerBlock, 0, s>>>(
        q,
        params,
        d_phaseInds,
        i
      );
      checkCudaErrors(hipFreeAsync(d_phaseInds, s));
    };
    memopt_adapter::registerAndExecuteTask(
      {i},
      {i},
      task,
      qureg,
      stream
    );
  }
}

__global__ void statevec_swapQubitAmpsBothLocalKernel(Qureg qureg, StateVecIndex_t globalIndex, int qb1, int qb2) {
  const StateVecIndex_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const StateVecIndex_t numTasks = qureg.numAmpsPerShard >> 2;
  if (idx >= numTasks) return;

  qreal* reVec = qureg.deviceStateVecShards[globalIndex].real;
  qreal* imVec = qureg.deviceStateVecShards[globalIndex].imag;

  StateVecIndex_t ind00, ind01, ind10;
  qreal re01, re10, im01, im10;

  ind00 = insertTwoZeroBits(idx, qb1, qb2);
  ind01 = flipBit(ind00, qb1);
  ind10 = flipBit(ind00, qb2);

  re01 = reVec[ind01];
  im01 = imVec[ind01];
  re10 = reVec[ind10];
  im10 = imVec[ind10];

  reVec[ind01] = re10;
  imVec[ind01] = im10;
  reVec[ind10] = re01;
  imVec[ind10] = im01;
}

__global__ void statevec_swapQubitAmpsOneLocalOneGlobalKernel(Qureg qureg, StateVecIndex_t globalIndex, int qb1, int qb2) {
  const StateVecIndex_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const StateVecIndex_t numTasks = qureg.numAmpsPerShard >> 1;
  if (idx >= numTasks) return;

  StateVecIndex_t globalIndexUp, globalIndexLo;
  StateVecIndex_t localIndexUp, localIndexLo;
  qreal re01, re10, im01, im10;

  globalIndexUp = globalIndex;
  globalIndexLo = flipBit(globalIndexUp, qb2 - qureg.numLocalBits);

  localIndexUp = insertZeroBit(idx, qb1);
  localIndexLo = flipBit(localIndexUp, qb1);

  re01 = qureg.deviceStateVecShards[globalIndexUp].real[localIndexLo];
  im01 = qureg.deviceStateVecShards[globalIndexUp].imag[localIndexLo];
  re10 = qureg.deviceStateVecShards[globalIndexLo].real[localIndexUp];
  im10 = qureg.deviceStateVecShards[globalIndexLo].imag[localIndexUp];

  qureg.deviceStateVecShards[globalIndexUp].real[localIndexLo] = re10;
  qureg.deviceStateVecShards[globalIndexUp].imag[localIndexLo] = im10;
  qureg.deviceStateVecShards[globalIndexLo].real[localIndexUp] = re01;
  qureg.deviceStateVecShards[globalIndexLo].imag[localIndexUp] = im01;
}

void memopt_statevec_swapQubitAmps(hipStream_t stream, Qureg qureg, int qb1, int qb2) {
  // Make sure qb1 < qb2
  if (qb2 < qb1) {
    int temp = qb1;
    qb1 = qb2;
    qb2 = temp;
  }

  if (qb2 < qureg.numLocalBits) {
    // Both are local bits
    StateVecIndex_t numThreadsPerBlock, numBlocks;
    numThreadsPerBlock = 128;
    numBlocks = ((qureg.numAmpsPerShard >> 2) + numThreadsPerBlock - 1) / numThreadsPerBlock;

    for (StateVecIndex_t i = 0; i < qureg.numShards; i++) {
      memopt_adapter::Task task = [=](Qureg q, hipStream_t s) {
        statevec_swapQubitAmpsBothLocalKernel<<<numBlocks, numThreadsPerBlock, 0, s>>>(
          q,
          i,
          qb1,
          qb2
        );
      };
      memopt_adapter::registerAndExecuteTask(
        {i},
        {i},
        task,
        qureg,
        stream
      );
    }
  } else if (qb1 < qureg.numLocalBits) {
    // qb1 is local bit while qb2 is global bit
    StateVecIndex_t numThreadsPerBlock, numBlocks;
    numThreadsPerBlock = 128;
    numBlocks = ((qureg.numAmpsPerShard >> 1) + numThreadsPerBlock - 1) / numThreadsPerBlock;

    for (StateVecIndex_t i = 0; i < (qureg.numShards >> 1); i++) {
      StateVecIndex_t globalIndexUp = insertZeroBit(i, qb2 - qureg.numLocalBits);
      StateVecIndex_t globalIndexLo = flipBit(globalIndexUp, qb2 - qureg.numLocalBits);
      memopt_adapter::Task task = [=](Qureg q, hipStream_t s) {
        statevec_swapQubitAmpsOneLocalOneGlobalKernel<<<numBlocks, numThreadsPerBlock, 0, s>>>(
          q,
          globalIndexUp,
          qb1,
          qb2
        );
      };
      memopt_adapter::registerAndExecuteTask(
        {globalIndexUp, globalIndexLo},
        {globalIndexUp, globalIndexLo},
        task,
        qureg,
        stream
      );
    }
  } else {
    // Both are global bits
    for (StateVecIndex_t i = 0; i < (qureg.numShards >> 2); i++) {
      StateVecIndex_t globalIndex00 = insertTwoZeroBits(i, qb1 - qureg.numLocalBits, qb2 - qureg.numLocalBits);
      StateVecIndex_t globalIndex01 = flipBit(globalIndex00, qb1 - qureg.numLocalBits);
      StateVecIndex_t globalIndex10 = flipBit(globalIndex00, qb2 - qureg.numLocalBits);

      memopt_adapter::Task task = [=](Qureg q, hipStream_t s) {
        qreal *realTemp, *imagTemp;
        checkCudaErrors(hipMallocAsync(&realTemp, q.numAmpsPerShard * sizeof(qreal), s));
        checkCudaErrors(hipMallocAsync(&imagTemp, q.numAmpsPerShard * sizeof(qreal), s));

        checkCudaErrors(hipMemcpyAsync(realTemp, q.deviceStateVecShards[globalIndex01].real, q.numAmpsPerShard * sizeof(qreal), hipMemcpyDefault, s));
        checkCudaErrors(hipMemcpyAsync(imagTemp, q.deviceStateVecShards[globalIndex01].imag, q.numAmpsPerShard * sizeof(qreal), hipMemcpyDefault, s));
        checkCudaErrors(hipMemcpyAsync(q.deviceStateVecShards[globalIndex01].real, q.deviceStateVecShards[globalIndex10].real, q.numAmpsPerShard * sizeof(qreal), hipMemcpyDefault, s));
        checkCudaErrors(hipMemcpyAsync(q.deviceStateVecShards[globalIndex01].imag, q.deviceStateVecShards[globalIndex10].imag, q.numAmpsPerShard * sizeof(qreal), hipMemcpyDefault, s));
        checkCudaErrors(hipMemcpyAsync(q.deviceStateVecShards[globalIndex10].real, realTemp, q.numAmpsPerShard * sizeof(qreal), hipMemcpyDefault, s));
        checkCudaErrors(hipMemcpyAsync(q.deviceStateVecShards[globalIndex10].imag, imagTemp, q.numAmpsPerShard * sizeof(qreal), hipMemcpyDefault, s));

        checkCudaErrors(hipFreeAsync(realTemp, s));
        checkCudaErrors(hipFreeAsync(imagTemp, s));
      };
      memopt_adapter::registerAndExecuteTask(
        {globalIndex01, globalIndex10},
        {globalIndex01, globalIndex10},
        task,
        qureg,
        stream
      );
    }
  }
}

hipGraph_t captureCudaGraphForFullQFT(hipStream_t stream, Qureg qureg) {
  // Does not support density matrix
  assert(!qureg.isDensityMatrix);

  checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

  // Start with top/left-most qubit, work down
  for (int q = qureg.numQubitsInStateVec - 1; q >= 0; q--) {
    memopt_statevec_hadamard(stream, qureg, q);

    if (q == 0)
      break;

    ApplyParamNamedPhaseFuncOverridesParams params;
    params.numRegs = 2;
    params.numQubitsPerReg[0] = q;
    params.numQubitsPerReg[1] = 1;
    for (int i = 0; i < q + 1; i++)
      params.qubits[i] = i;

    params.numParams = 1;
    params.params[1] = M_PI / (1 << q);

    params.encoding = UNSIGNED;
    params.phaseFuncName = SCALED_PRODUCT;

    params.numOverrides = 0;

    params.conj = 0;

    memopt_statevec_applyParamNamedPhaseFuncOverrides(stream, qureg, params);
  }

  for (int i = 0; i < (qureg.numQubitsInStateVec / 2); i++) {
    int qb1 = i;
    int qb2 = qureg.numQubitsInStateVec - i - 1;

    memopt_statevec_swapQubitAmps(stream, qureg, qb1, qb2);
  }

  hipGraph_t graph;
  checkCudaErrors(hipStreamEndCapture(stream, &graph));
  return graph;
}

#ifdef __cplusplus
extern "C" {
#endif

#define NUM_GLOBAL_BITS 4

// Copied from QuEST_gpu_common.cu
int GPUExists(void) {
  int deviceCount, device;
  int gpuDeviceCount = 0;
  struct hipDeviceProp_t properties;
  hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
  if (cudaResultCode != hipSuccess) deviceCount = 0;
  /* machines with no GPUs can still report one emulation device */
  for (device = 0; device < deviceCount; ++device) {
    hipGetDeviceProperties(&properties, device);
    if (properties.major != 9999) { /* 9999 means emulation only */
      ++gpuDeviceCount;
    }
  }
  if (gpuDeviceCount)
    return 1;
  else
    return 0;
}

// Copied from QuEST_gpu_common.cu
void seedQuEST(QuESTEnv* env, unsigned long int* seedArray, int numSeeds) {
  // free existing seed array, if exists
  if (env->seeds != NULL)
    free(env->seeds);

  // record keys in permanent heap
  env->seeds = (unsigned long int*)malloc(numSeeds * sizeof *(env->seeds));
  for (int i = 0; i < numSeeds; i++)
    (env->seeds)[i] = seedArray[i];
  env->numSeeds = numSeeds;

  // pass keys to Mersenne Twister seeder
  init_by_array(seedArray, numSeeds);
}

QuESTEnv createQuESTEnv() {
  // Initialize memopt
  memopt::ConfigurationManager::exportDefaultConfiguration();
  memopt::ConfigurationManager::loadConfiguration();

  validateGPUExists(GPUExists(), __func__);

  QuESTEnv env;
  env.rank = 0;
  env.numRanks = 1;

  env.seeds = NULL;
  env.numSeeds = 0;
  seedQuESTDefault(&env);

  return env;
}

void destroyQuESTEnv(QuESTEnv env) {
  free(env.seeds);
}

void statevec_createQureg(Qureg* qureg, int numQubits, QuESTEnv env) {
  assert((1 << NUM_GLOBAL_BITS) <= MAX_NUM_SHARDS);
  assert(numQubits > NUM_GLOBAL_BITS);

  const StateVecIndex_t numShards = 1L << NUM_GLOBAL_BITS;
  const StateVecIndex_t numTotalAmps = 1L << numQubits;
  const StateVecIndex_t numAmpsPerShard = numTotalAmps / numShards;

  qureg->numQubitsInStateVec = numQubits;
  qureg->numAmpsPerChunk = numTotalAmps;
  qureg->numAmpsPerShard = numAmpsPerShard;
  qureg->numAmpsTotal = numTotalAmps;
  qureg->chunkId = env.rank;
  qureg->numChunks = env.numRanks;
  qureg->numShards = numShards;
  qureg->numGlobalBits = NUM_GLOBAL_BITS;
  qureg->numLocalBits = numQubits - NUM_GLOBAL_BITS;
  qureg->isDensityMatrix = 0;

  for (StateVecIndex_t i = 0; i < qureg->numShards; i++) {
    memopt_adapter::allocateShardAndRegister(&(qureg->deviceStateVecShards[i].real), qureg->numAmpsPerShard * sizeof(qreal));
    memopt_adapter::allocateShardAndRegister(&(qureg->deviceStateVecShards[i].imag), qureg->numAmpsPerShard * sizeof(qreal));
  }
}

void statevec_destroyQureg(Qureg qureg, QuESTEnv env) {
  for (StateVecIndex_t i = 0; i < qureg.numShards; i++) {
    checkCudaErrors(hipFree(qureg.deviceStateVecShards[i].real));
    checkCudaErrors(hipFree(qureg.deviceStateVecShards[i].imag));
  }
}

void applyFullQFTWithMemopt(Qureg qureg) {
  size_t totalShardSize = 0;
  for (const auto& [addr, size] : memopt::MemoryManager::managedMemoryAddressToSizeMap) {
    totalShardSize += size;
  }
  printf("totalShardSize (MiB) = %.6lf\n", (double)totalShardSize * 1e-6);

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  hipGraph_t graph = captureCudaGraphForFullQFT(stream, qureg);

  checkCudaErrors(hipGraphDebugDotPrint(graph, "graph.dot", hipGraphDebugDotFlagsVerbose));

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph));
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(stream));
}

#ifdef __cplusplus
}
#endif
